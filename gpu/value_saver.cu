#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include "value_saver.h"
#include "mytools.h"
using namespace std;
typedef unsigned long long ull;

value_saver::value_saver() {
    hipHostAlloc(&vals, MAX_SIZ, hipHostMallocMapped);
    hipSetDevice(1);
    hipMalloc(&d_vals, DEV_SIZ);
    hipSetDevice(0);
    valptr = vals; d_valptr = d_vals;
    offset = 0; flg = -1;
}

void value_saver::save(size_t siz, const vec *v, int typ) {
    size_t cnt = siz/sizeof(vec);
    if (!~flg && valptr+cnt > vals+MAX_CNT) {
        flg = offset;
    }
    if (!~flg) {
        if (typ == 0) {
            memcpy(valptr, v, siz);
        } else {
            hipMemcpy(valptr, v, siz, hipMemcpyDeviceToHost);
        }
        valptr += cnt;
    } else {
        if (typ == 0) {
            hipMemcpy(d_valptr, v, siz, hipMemcpyHostToDevice);
            d_valptr += cnt;
        } else {
            hipMemcpyPeer(d_valptr, 1, v, 0, siz);
            d_valptr += cnt;
        }
    }
    offset += cnt;
}

size_t value_saver::qoffset() {
    return offset;
}

size_t value_saver::onHost() {
    return flg;
}

vec* value_saver::hostData() {
    return vals;
}

vec* value_saver::deviceData() {
    return d_vals;
}