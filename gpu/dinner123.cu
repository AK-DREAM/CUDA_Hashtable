#include "hip/hip_runtime.h"
#include <utility>
#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <string>
#include "mytools.h"
#include "dinner123.h"
using namespace std;
typedef unsigned long long ull;

dinner123::dinner123() {
    hipMalloc(&Keys, sizeof(ull)<<TSIZ);
    hipMalloc(&Vals, sizeof(ull)<<TSIZ);
    hipHostAlloc(&Exi, 1ull<<30, hipHostMallocMapped);
    hipHostAlloc(&Ans, 1ull<<30, hipHostMallocMapped); 
    insTime = fndTime = 0;
}

#define mix(h) ({					\
			(h) ^= (h) >> 23;		\
			(h) *= 0x2127599bf4325c37ULL;	\
			(h) ^= (h) >> 47; })

__device__ ull hash_val(ull v, ull seed=114514) {
	const ull m = 0x880355f21e6d1965ULL;
	ull h = seed;
	h ^= mix(v);
	h *= m;
    return mix(h)&((1ull<<TSIZ)-1);
}

__device__ void insert1(ull *Keys, ull *Vals, ull key, ull ptr) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            atomicMax(&Vals[hs], ptr);
            return;
        }
        hs = (hs+1)&((1ull<<TSIZ)-1);
    }
}
__global__ void insert_kernel(ull *Keys, ull *Vals, const ull *d_keys, ull offset, size_t cnt) {
    LOOP(i, cnt) {
        insert1(Keys, Vals, d_keys[i], offset+i);
    }
}
void dinner123::insert(size_t cnt, const ull* keys, const vec* values, hipStream_t stream = 0) {
    ull offset = saver.qoffset();
    saver.save(cnt*sizeof(vec), values, 1);
    insert_kernel<<<80,1024>>>(Keys, Vals, keys, offset, cnt);
    hipDeviceSynchronize();
}
void dinner123::Insert(const char *keyfile, const char *valfile) {
    size_t cnt = loader.load_keyfile(keyfile); 
    ull offset = saver.qoffset();
    size_t siz = loader.load_valfile(valfile);

    auto exec = [&]() {
        saver.save(siz, loader.valdata(), 0);
        insert_kernel<<<80,1024>>>(Keys, Vals, loader.keydata(), offset, cnt);
    };
    PERF_GPU(insTime, exec(););
    hipDeviceSynchronize();

    puts(~saver.onHost()?"Device":"Host");
}

__device__ ull query1(ull *Keys, ull *Vals, ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return (1ull<<63);
        hs = (hs+1)&((1u<<TSIZ)-1);
    }
}
__global__ void query_kernel(ull *Keys, ull *Vals, const ull *d_keys, ull *Ptr, bool *Exi, size_t cnt) {
    LOOP(i, cnt) {
        Ptr[i] = query1(Keys, Vals, d_keys[i]);
        Exi[i] = !(Ptr[i]>>63&1);
    }
}

__global__ void findVal_kernel0(vec *data, const ull *Ptr, vec *Ans, size_t cnt, size_t flg) {
    LOOP(i, cnt<<5) {
        int id = i>>5, j = i&31;
        if ((Ptr[id]>>63&1) || Ptr[id] >= flg) continue;
        float *pt1 = (float*)(Ans+id), *pt2 = (float*)(data+Ptr[id]);
        *(pt1+j) = *(pt2+j);
        *(pt1+j+32) = *(pt2+j+32);
    }
}

__global__ void findVal_kernel1(vec *data, ull *Ptr, vec *Ans, size_t cnt, size_t flg) {
    LOOP(i, cnt<<5) {
        int id = i>>5, j = i&31;
        if ((Ptr[id]>>63&1) || Ptr[id] < flg) continue;
        float *pt1 = (float*)(Ans+id), *pt2 = (float*)(data+Ptr[id]);
        *(pt1+j) = *(pt2+j);
        *(pt1+j+32) = *(pt2+j+32);
    }
}

void dinner123::find(size_t cnt, const ull* d_keys, vec* Ans, bool* Exi, hipStream_t stream = 0) {
    ull *Ptr; hipMalloc(&Ptr, cnt*sizeof(ull));
    query_kernel<<<80,1024>>>(Keys, Vals, d_keys, Ptr, Exi, cnt);
    hipDeviceSynchronize();
    size_t flg = saver.onHost();
    findVal_kernel0<<<80,1024>>>(saver.hostData(), Ptr, Ans, cnt, flg);
    if (~flg) {
        hipSetDevice(1);
        ull *Ptr1; hipMalloc(&Ptr1, cnt*sizeof(ull));
        vec *Ans1; hipMalloc(&Ans1, cnt*sizeof(vec));
        hipMemcpyPeer(Ptr1, 1, Ptr, 0, cnt*sizeof(ull));
        findVal_kernel1<<<80,1024>>>(saver.deviceData(), Ptr1, Ans1, cnt, flg);
        hipMemcpy(Ans, Ans1, cnt*sizeof(vec), hipMemcpyDeviceToHost);
        hipSetDevice(0);
    }
    hipDeviceSynchronize();
}

string changeExtension(const char *file, string ext) {
    string str(file);
    size_t pos = str.rfind('.');
    str.replace(pos, std::string::npos, ext);
    return str.c_str();
}

void dinner123::Find(const char *keyfile) {
    size_t cnt = loader.load_keyfile(keyfile); 
    auto exec = [&]() {
        find(cnt, loader.keydata(), Ans, Exi);
    };
    PERF_GPU(fndTime, exec(););
    FILE *OUT0 = fopen(changeExtension(keyfile, ".myvals").c_str(), "wb");
    fwrite(Ans, 1, cnt*sizeof(vec), OUT0);
    fclose(OUT0); 

    FILE *OUT1 = fopen(changeExtension(keyfile, ".exists").c_str(), "wb");
    fwrite(Exi, 1, cnt*sizeof(bool), OUT1);
    fclose(OUT1);
}
