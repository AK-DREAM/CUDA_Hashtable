#include "hip/hip_runtime.h"
#include <utility>
#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
#include "dinner123.h"
using namespace std;
typedef unsigned long long ull;

#define mix(h) ({					\
			(h) ^= (h) >> 23;		\
			(h) *= 0x2127599bf4325c37ULL;	\
			(h) ^= (h) >> 47; })

__device__ ull hash_val(ull v, ull seed=114514) {
	const ull m = 0x880355f21e6d1965ULL;
	ull h = seed;
	h ^= mix(v);
	h *= m;
    return mix(h)&((1u<<TSIZ)-1);
}

__device__ void insert1(ull *Keys, ull *Vals, ull key, ull ptr) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            atomicMax(&Vals[hs], ptr);
            return;
        }
        hs = (hs+1)&((1u<<TSIZ)-1);
    }
}

__global__ void insert_kernel(ull *Keys, ull *Vals, ull *d_keys, ull d_ptr, int cnt) {
    LOOP(i, cnt) {
        insert1(Keys, Vals, d_keys[i], d_ptr+i*sizeof(vec));
    }
}

dinner123::dinner123() {
    hipMalloc(&Keys, sizeof(ull)<<TSIZ);
    hipMalloc(&Vals, sizeof(ull)<<TSIZ);
}

void dinner123::insert(size_t cnt, ull *h_keys, ull h_ptr) {
    ull *d_keys; 
    d_keys = h_keys;
    // hipMalloc(&d_keys, sizeof(ull)*cnt);
    // hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);

    auto work = [&]() {
        insert_kernel<<<80,1024>>>(Keys, Vals, d_keys, h_ptr, cnt);
        hipDeviceSynchronize();
    };
    work();
    // hipFree(d_keys); 
}

__device__ ull query1(ull *Keys, ull *Vals, ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return NULL;
        hs = (hs+1)&((1u<<TSIZ)-1);
    }
}
__global__ void query_kernel(ull *Keys, ull *Vals, ull *d_keys, ull *d_ans, bool *d_ok, int cnt) {
    LOOP(i, cnt) {
        d_ans[i] = query1(Keys, Vals, d_keys[i]);
        d_ok[i] = !!d_ans[i];
    }
}
void dinner123::query(size_t cnt, ull *h_keys, ull *h_ans, bool *h_ok) {
    //reverse(h_keys, h_keys+cnt);
    ull *d_keys, *d_ans; bool *d_ok;
    d_keys = h_keys; d_ans = h_ans; d_ok = h_ok;
    // hipHostGetDevicePointer(&d_keys, h_keys, 0);
    // hipHostGetDevicePointer(&d_ans, h_ans, 0);
    // hipHostGetDevicePointer(&d_ok, h_ok, 0);
    // hipMalloc(&d_keys, sizeof(ull)*cnt);
    // hipMalloc(&d_ans, sizeof(ull)*cnt);
    // hipMalloc(&d_ok, sizeof(bool)*cnt);
    // hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);

    auto work = [&]() {
        query_kernel<<<80,1024>>>(Keys, Vals, d_keys, d_ans, d_ok, cnt);
        hipDeviceSynchronize();
    };
    work();
    
    // hipMemcpy(h_ans, d_ans, sizeof(ull)*cnt, hipMemcpyDeviceToHost);
    // hipMemcpy(h_ok, d_ok, sizeof(bool)*cnt, hipMemcpyDeviceToHost);
    // hipFree(d_keys); 
    // hipFree(d_ans); 
    // hipFree(d_ok);
}