#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
#include "dinner123.h"
using namespace std;
typedef unsigned long long ull;

int n = 100;
int pt[] = {0,10,20,50,100};

int main() {
    dinner123 Hashtable;
    cerr << "Initialized\n";
    for (int t = 0; t < 4; t++) {
        for (int i = pt[t]; i < pt[t+1]; i++) {
            cerr << "insert: " << i << endl;
            string name = "data/A/part_"+to_string(i);
            Hashtable.Insert((name+".keys").c_str(), (name+".vals").c_str());
            // cerr << "insert: " << i << " OK\n";
        }
        for (int i = pt[t]; i < pt[t+1]; i++) {
            cerr << "find: " << i << endl;
            string name = "data/A/part_"+to_string(i);
            Hashtable.Find((name+".keys").c_str());
            // cerr << "find: " << i << " OK\n";
        }
    }
    // Hashtable.Find("data/A/part_50.keys");

    for (int i = 0; i < 100; i++) {
        string name = "data/A/part_"+to_string(i);
        system(("diff "+name+".vals "+name+".myvals").c_str());
    }

    fprintf(stderr, "Insert Time: %f\nFind Time: %f\n", Hashtable.insTime, Hashtable.fndTime);
    return 0;
}