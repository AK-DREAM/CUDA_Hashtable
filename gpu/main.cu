#include "hip/hip_runtime.h"
#include <cstdio>
#include "data_loader.h"
#include "mytools.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;
typedef unsigned long long ull;
typedef vec<64> T;

const int K = 30;

__device__ unsigned hash_val(ull key) {
    return key&((1<<K)-1);
}

__device__ void insert1(ull *Keys, ull *Vals, ull key, ull ptr) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            Vals[hs] = ptr;
            return;
        }
        hs = (hs+1)&((1<<K)-1);
    }
}

__global__ void insert_kernel(ull *Keys, ull *Vals, ull *d_keys, ull *d_ptr, int cnt) {
    LOOP(i, cnt) {
        insert1(Keys, Vals, d_keys[i], *d_ptr+i*sizeof(T));
    }
}

void insert(ull *Keys, ull *Vals, ull *h_keys, T *h_ptr, int cnt) {
    ull *d_keys; ull *d_ptr;
    ull num = (ull)h_ptr;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ptr, sizeof(ull));
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, &num, sizeof(ull), hipMemcpyHostToDevice);  

    auto work = [&]() {
        insert_kernel<<<64,64>>>(Keys, Vals, d_keys, d_ptr, cnt);
        hipDeviceSynchronize();
    };
    work();
    
    hipFree(d_keys); hipFree(d_ptr);
}

__device__ ull query1(ull *Keys, ull *Vals, ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return NULL;
        hs = (hs+1)&((1<<K)-1);
    }
}
__global__ void query_kernel(ull *Keys, ull *Vals, ull *d_keys, ull *d_ans, int cnt) {
    LOOP(i, cnt) {
        d_ans[i] = query1(Keys, Vals, d_keys[i]);
    }
}
void query(ull *Keys, ull *Vals, ull *h_keys, ull *h_ans, int cnt) {
    reverse(h_keys, h_keys+cnt);
    ull *d_keys; ull *d_ans;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ans, sizeof(ull)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);

    auto work = [&]() {
        query_kernel<<<64,64>>>(Keys, Vals, d_keys, d_ans, cnt);
        hipDeviceSynchronize();
    };
    work();
    
    hipMemcpy(h_ans, d_ans, sizeof(ull)*cnt, hipMemcpyDeviceToHost);
    hipFree(d_keys); hipFree(d_ans);
}

int main() {
    data_loader<ull> data_Keys("../data/my2.keys");
    data_loader<T> data_Vals("../data/my2.vals");
    ull *Keys, *Vals;
    hipMalloc(&Keys, sizeof(ull)<<K);
    hipMalloc(&Vals, sizeof(ull)<<K);
    ull *Ans = (ull*)malloc(data_Keys.count()*sizeof(ull));

    PERF_GPU(
        insert(Keys, Vals, data_Keys.data(), data_Vals.data(), data_Keys.count());
    );
    PERF_GPU(
        query(Keys, Vals, data_Keys.data(), Ans, data_Keys.count());
    );
    
    int cnt = data_Keys.count();
    printf("%d\n", cnt);
    // for (int i = 0; i < cnt; i++) if (Ans[i]) {
    //     for (int j = 0; j < 64; j++) printf("%f ", ((T*)Ans[i])->v[j]);
    //     puts("");
    // }
    return 0;
}