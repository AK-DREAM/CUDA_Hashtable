#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
#include "dinner123.h"
using namespace std;
typedef unsigned long long ull;

int main() {
    data_loader<ull> data_Keys("../data/my2.keys");
    data_loader<vec> data_Vals("../data/my2.vals");

    dinner123 Hashtable;
    ull *Ans; // = (ull*)malloc(data_Keys.count()*sizeof(ull));
    // hipHostMalloc(&Ans, data_Keys.count()*sizeof(ull));
    hipHostAlloc(&Ans, data_Keys.count()*sizeof(ull), hipHostMallocMapped);
    bool *Ok; // = (bool*)malloc(data_Keys.count()*sizeof(bool));; 
    // hipHostMalloc(&Ok, data_Keys.count()*sizeof(bool));
    hipHostAlloc(&Ok, data_Keys.count()*sizeof(bool), hipHostMallocMapped);

    PERF_GPU(
        Hashtable.insert(data_Keys.count(), data_Keys.data(), (ull)data_Vals.data());
    );
    CUDA_CHECK_ERROR();
    PERF_GPU(
        Hashtable.query(data_Keys.count(), data_Keys.data(), Ans, Ok);
    );
    CUDA_CHECK_ERROR();
    
    int cnt = data_Keys.count();
    printf("%d\n", cnt);
    for (int i = 0; i < 1; i++) if (Ok[i]) {
        for (int j = 0; j < 64; j++) printf("%f ", ((vec*)Ans[i])->v[j]);
        puts("");
    }
    hipHostFree(Ans);
    hipHostFree(Ok);
    return 0;
}