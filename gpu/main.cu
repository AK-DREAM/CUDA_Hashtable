#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
using namespace std;
typedef unsigned long long ull;

const int K = 30;

#define mix(h) ({					\
			(h) ^= (h) >> 23;		\
			(h) *= 0x2127599bf4325c37ULL;	\
			(h) ^= (h) >> 47; })

__device__ ull hash_val(ull v, ull seed=114514) {
	const ull m = 0x880355f21e6d1965ULL;
	ull h = seed;
	h ^= mix(v);
	h *= m;
    return mix(h)&((1u<<K)-1);
}

__device__ void insert1(ull *Keys, ull *Vals, ull key, ull ptr) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            Vals[hs] = ptr;
            return;
        }
        hs = (hs+1)&((1u<<K)-1);
    }
}

__global__ void insert_kernel(ull *Keys, ull *Vals, ull *d_keys, ull d_ptr, int cnt) {
    LOOP(i, cnt) {
        insert1(Keys, Vals, d_keys[i], d_ptr+i*sizeof(TP));
    }
}

void insert(ull *Keys, ull *Vals, ull *h_keys, TP *h_ptr, int cnt) {
    ull *d_keys; 
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);

    auto work = [&]() {
        insert_kernel<<<80,64>>>(Keys, Vals, d_keys, (ull)h_ptr, cnt);
        hipDeviceSynchronize();
    };
    work();
    hipFree(d_keys); 
}

__device__ ull query1(ull *Keys, ull *Vals, ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return NULL;
        hs = (hs+1)&((1u<<K)-1);
    }
}
__global__ void query_kernel(ull *Keys, ull *Vals, ull *d_keys, ull *d_ans, int cnt) {
    LOOP(i, cnt) {
        d_ans[i] = query1(Keys, Vals, d_keys[i]);
    }
}
void query(ull *Keys, ull *Vals, ull *h_keys, ull *h_ans, int cnt) {
    // reverse(h_keys, h_keys+cnt);
    ull *d_keys; ull *d_ans;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ans, sizeof(ull)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);

    auto work = [&]() {
        query_kernel<<<80,64>>>(Keys, Vals, d_keys, d_ans, cnt);
        hipDeviceSynchronize();
    };
    work();
    
    hipMemcpy(h_ans, d_ans, sizeof(ull)*cnt, hipMemcpyDeviceToHost);
    hipFree(d_keys); hipFree(d_ans);
}

int main() {
    data_loader<ull> data_Keys("../data/part_0.keys");
    data_loader<TP> data_Vals("../data/part_0.vals");
    puts("OK");
    ull *Keys, *Vals;
    hipMalloc(&Keys, sizeof(ull)<<K);
    hipMalloc(&Vals, sizeof(ull)<<K);
    ull *Ans = (ull*)malloc(data_Keys.count()*sizeof(ull));

    PERF_GPU(
        insert(Keys, Vals, data_Keys.data(), data_Vals.data(), data_Keys.count());
    );
    PERF_GPU(
        query(Keys, Vals, data_Keys.data(), Ans, data_Keys.count());
    );
    
    int cnt = data_Keys.count();
    printf("%d\n", cnt);
    for (int i = 0; i < 100; i++) if (Ans[i]) {
        for (int j = 0; j < 64; j++) printf("%f ", ((TP*)Ans[i])->v[j]);
        puts("");
    }
    return 0;
}