#include "hip/hip_runtime.h"
#include "reader.h"
#include "../dinner123.h"
#include "cpu.h"
#include <bits/stdc++.h>
using namespace std;
typedef unsigned long long uint64;
template <typename T>
double CPU_PERF(T func) {
	auto t0 = chrono::high_resolution_clock::now();
	func();
	auto t1 = chrono::high_resolution_clock::now();
	double duration = chrono::duration<double>(t1 - t0).count();
	return duration;
}
CPUHashTable<ull, float, 8> cpuhstb;
template <typename T>
double GPU_PERF(T func) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	func();
	hipEventRecord(stop);
	hipError_t err = hipStreamSynchronize(0);
	if (err != hipSuccess) {
		cerr << "PERF_DEV run failed.\n";
		exit(0);
	}
	float duration = 0;
	hipEventElapsedTime(&duration, start, stop);
	return duration / 1000;
}
int main() {
	system("./gen");
	cerr << "Finish generating\n";
	data_loader insertion("input1");
	file_loader<uint64> finding("input2.keys");
	dinner123 Hashtable;
	int n = insertion.size();
	int a[5] = {0, 0.1*n, 0.2*n, 0.5*n, n};
	double alpha[5] = {0, 0.28, 0.24, 0.18, 0.1};
	double beta[5] = {0, 0.42, 0.56, 0.72, 0.9};
	cerr << insertion.size() * insertion.dim() * sizeof(uint64) << '\n';
	uint64 *answer_cpu = (uint64 *)malloc(insertion.size() * insertion.dim() * sizeof(uint64)), *answer_gpu = (uint64 *)malloc(insertion.size() * insertion.dim() * sizeof(uint64));
	bool *exist_cpu = (bool *)malloc(insertion.size()), *exist_gpu = (bool *)malloc(insertion.size());
	double cpu_total_score = 0, gpu_total_score = 0;
	// cerr << "CPU PERF-----------------------------------------\n";
	// for (int i = 1; i <= 4; ++i) {
	// 	if (a[i] == a[i - 1])
	// 		continue;
	// 	double cpu_insert_time = CPU_PERF([&] { cpuhstb.insert(a[i] - a[i - 1], insertion.keys() + a[i - 1], insertion.vals() + a[i - 1] * insertion.dim()); });
	// 	double cpu_insert_qps = (a[i] - a[i - 1]) / cpu_insert_time;
	// 	double cpu_find_time = CPU_PERF([&] { cpuhstb.find(a[i] - a[i - 1], finding.data() + a[i - 1], answer_cpu + a[i - 1] * insertion.dim(), exist_cpu + a[i - 1]); });
	// 	double cpu_find_qps = (a[i] - a[i - 1]) / cpu_find_time;
	// 	int rate = a[i] * 100. / n + 0.5;
	// 	double score = (alpha[i] * cpu_find_qps + beta[i] * cpu_insert_qps) * (1 - exp(-exp(1)));
	// 	cpu_total_score += score;
	// 	cerr << "Finishing " << rate << "%, score=" << score << ", insert_qps=" << cpu_insert_qps << ", find_qps=" << cpu_find_qps << "\n";
	// }
	cerr << "GPU PERF-----------------------------------------\n";
	for (int i = 1; i <= 4; ++i) {
		if (a[i] == a[i - 1])
			continue;
		double gpu_insert_time = GPU_PERF([&] { Hashtable.insert(a[i] - a[i - 1], insertion.keys() + a[i - 1], (unsigned long long)(insertion.vals() + a[i - 1] * insertion.dim())); });
		double gpu_insert_qps = (a[i] - a[i - 1]) / gpu_insert_time;
		double gpu_find_time = GPU_PERF([&] { Hashtable.query(a[i] - a[i - 1], finding.data() + a[i - 1], answer_gpu + a[i - 1] * insertion.dim(), exist_gpu + a[i - 1]); });
		double gpu_find_qps = (a[i] - a[i - 1]) / gpu_find_time;
		int cnt = 0;
		for (int j = a[i - 1]; j < a[i]; ++j) {
			if (exist_cpu[j] == exist_gpu[j]) {
				cnt += answer_cpu[j] == answer_gpu[j] || !exist_cpu[j];
			} else
				cerr << "zzh\n";
		}
		double accuracy = 1.0 * cnt / (a[i] - a[i - 1]);
		int rate = a[i] * 100. / n + 0.5;
		double score = (alpha[i] * gpu_find_qps + beta[i] * gpu_insert_qps) * (1 - exp(-exp(1) * accuracy));
		gpu_total_score += score;
		cerr << "Finishing " << rate << "%, score=" << score << ", insert_qps=" << gpu_insert_qps << ", find_qps=" << gpu_find_qps << " ,accuracy=" << accuracy << " (" << cnt << "/" << a[i] - a[i - 1]
			 << ")\n";
	}
	cerr << "CPU Total Score: " << cpu_total_score << '\n';
	cerr << "GPU Total Score: " << gpu_total_score << '\n';
	return 0;
}