#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define CUDA_CHECK_ERROR() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(-1); \
    } else { \
        std::cerr << "OK!\n"; \
    } \
}

__global__ void test_kernel(int cnt, int *a, int *b) {
    int i = threadIdx.x;
    if (i < 100) b[i] = *(a+i);
}

__global__ void print_kernel(int cnt, int *b) {
    int i = threadIdx.x;
    if (i < 100) printf("%d\n", b[i]);
}

int main() {
    int *a, *b;
    hipHostAlloc(&a, 100*sizeof(int), hipHostMallocMapped);
    hipMalloc(&b, 100*sizeof(int));
    for (int i = 0; i < 100; i++) a[i] = i;
    test_kernel<<<1,100>>>(100, a, b);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    print_kernel<<<1,100>>>(100, b);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    return 0;
}