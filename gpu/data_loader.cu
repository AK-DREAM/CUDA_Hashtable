#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
using namespace std;

template <typename T>
data_loader<T>::data_loader(const char *name) {
    FILE *IN = fopen(name, "rb");
    if (IN == nullptr) {
        puts("ERROR");
        assert(0);
    }
    fseek(IN, 0, SEEK_END);
    file_size = ftell(IN);
    rewind(IN);
    // buf = (T*)malloc(file_size);
    hipHostAlloc(&buf, file_size, hipHostMallocMapped);

    T* ptr = buf; long long res = file_size;
    while (res > 0) {
        size_t num = fread(ptr, 1, 1<<23, IN);
        ptr += num; res -= (1<<23);
        printf("%lld\n", res);
    }
    fclose(IN);
}

template <typename T>
data_loader<T>::~data_loader() {
    hipHostFree(buf);
}

template <typename T>
size_t data_loader<T>::count() {
    return file_size/sizeof(T);
}

template <typename T>
T* data_loader<T>::data() {
    return buf;
}

template class data_loader<unsigned long long>;
template class data_loader<vec>;