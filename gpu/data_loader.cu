#include <cstdio>
#include <cstdlib>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "data_loader.h"
#include "mytools.h"
#include "value_saver.h"
using namespace std;
typedef unsigned long long ull;

data_loader::data_loader() {
    hipHostAlloc(&keybuf, 2ull<<30, hipHostMallocMapped);
    hipHostAlloc(&valbuf, 2ull<<30, hipHostMallocMapped);
}

data_loader::~data_loader() {
    hipHostFree(keybuf);
    hipHostFree(valbuf);
}

size_t data_loader::load_keyfile(const char *name) {
    FILE *IN = fopen(name, "rb");
    fseek(IN, 0, SEEK_END);
    size_t file_size = ftell(IN);
    rewind(IN);

    ull *ptr = keybuf; ull res = 0;
    while (res < file_size) {
        size_t num = fread(ptr, 1, 1ull<<30, IN);
        ptr += num/sizeof(ull); res += num;
    }
    fclose(IN);
    return file_size/sizeof(ull);
}
size_t data_loader::load_valfile(const char *name) {
    FILE *IN = fopen(name, "rb");
    fseek(IN, 0, SEEK_END);
    size_t file_size = ftell(IN);
    rewind(IN);

    size_t siz = fread(valbuf, 1, 1<<30, IN);

    fclose(IN);
    return siz;
}

ull* data_loader::keydata() {
    return keybuf;
}

vec* data_loader::valdata() {
    return valbuf;
}