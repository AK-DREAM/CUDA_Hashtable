#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "dinner123.h"
using namespace std;
typedef unsigned long long ull;


#define LOOP(i, n) \
    for (size_t i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) 

template <typename T>
dinner123<T>::dinner123(int _K): K(_K) {
    hipMalloc(&Keys, sizeof(ull)<<K);
    hipMalloc(&Vals, sizeof(T)<<K);
}

template <typename T>
unsigned dinner123<T>::hash_val(ull key) {
    return key&((1<<K)-1);
}

template <typename T>
__device__ void dinner123<T>::insert1(ull key, T val) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            Vals[hs] = val;
            return;
        }
        now = (now+1)&((1<<K)-1);
    }
}

template <typename T>
__global__ void insert_kernel(ull *h_keys, T *h_vals, int cnt) {
    LOOP(i, cnt) {
        insert1(h_keys[i], h_vals[i]);
    }
}

template <typename T>
void dinner123<T>::insert(ull *h_keys, T *h_vals, int cnt) {
    ull *d_keys; T *d_vals;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_vals, sizeof(T)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_vals, h_vals, sizeof(T)*cnt, hipMemcpyHostToDevice);
    insert_kernel<<<64,1024>>><T>(d_keys, d_vals, cnt);
}