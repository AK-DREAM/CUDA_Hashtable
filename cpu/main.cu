#include <cstdio>
#include "data_loader.h"
#include "dinner123.h
#include <hip/hip_runtime.h>
using namespace std;
typedef unsigned long long ull;

const int HASH_SIZE = (1<<20);
const int N = 1000005;

int head[HASH_SIZE+5];
ull keys[N]; float vals[N];



int main() {
    data_loader<ull> Keys("../part_0.keys");
    data_loader<vec<64>> Values("../part_0.vals");
    
    table.insert(Keys.data(), Values.data(), Keys.count());
    return 0;
}