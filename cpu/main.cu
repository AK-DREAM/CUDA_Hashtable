#include "hip/hip_runtime.h"
#include <cstdio>
#include "data_loader.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;
typedef unsigned long long ull;

#define LOOP(i, n) \
    for (size_t i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) 

#define CUDA_CHECK_ERROR() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(-1); \
    } \
}

const int N = 1000005, K = 20;

__device__ ull Keys[1<<K]; 
__device__ vec<64> Vals[1<<K];
vec<64> Ans[N];

__device__ unsigned hash_val(ull key) {
    return key&((1<<K)-1);
}

__device__ void insert1(ull key, vec<64> val) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            Vals[hs] = val;
            return;
        }
        hs = (hs+1)&((1<<K)-1);
    }
}
__global__ void insert_kernel(ull *d_keys, vec<64> *d_vals, int cnt) {
    LOOP(i, cnt) {
        insert1(d_keys[i], d_vals[i]);
    }
}
__global__ void print_kernel() {
    printf("Hello");
}

void insert(ull *h_keys, vec<64> *h_vals, int cnt) {
    ull *d_keys; vec<64> *d_vals;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_vals, sizeof(vec<64>)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_vals, h_vals, sizeof(vec<64>)*cnt, hipMemcpyHostToDevice);
    insert_kernel<<<1,1>>>(d_keys, d_vals, cnt);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    hipFree(d_keys); hipFree(d_vals);
}

__device__ vec<64> query1(ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return Vals[0];
        hs = (hs+1)&((1<<K)-1);
    }
}
__global__ void query_kernel(ull *d_keys, vec<64> *d_ans, int cnt) {
    LOOP(i, cnt) {
        d_ans[i] = query1(d_keys[i]);
    }
}
void query(ull *h_keys, vec<64> *h_ans, int cnt) {
    reverse(h_keys, h_keys+cnt);
    ull *d_keys; vec<64> *d_ans;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ans, sizeof(vec<64>)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    query_kernel<<<64,64>>>(d_keys, d_ans, cnt);
    hipMemcpy(h_ans, d_ans, sizeof(vec<64>) *cnt, hipMemcpyDeviceToHost);
    hipFree(d_keys); hipFree(d_ans);
}

int main() {
    data_loader<ull> data_Keys("../part_0.keys");
    data_loader<vec<64>> data_Vals("../part_0.vals");
    insert(data_Keys.data(), data_Vals.data(), data_Keys.count());
    query(data_Keys.data(), Ans, data_Keys.count());
    int cnt = data_Keys.count();
    printf("%d\n", cnt);
    for (int i = 0; i < cnt; i++) {
        for (int j = 0; j < 64; j++) printf("%f ", Ans[i].v[j]);
        puts("");
    }
    return 0;
}