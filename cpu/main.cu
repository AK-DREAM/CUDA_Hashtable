#include "hip/hip_runtime.h"
#include <cstdio>
#include "data_loader.h"
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;
typedef unsigned long long ull;
typedef vec<64> T;

#define LOOP(i, n) \
    for (size_t i = threadIdx.x+blockIdx.x*blockDim.x; i < n; i += blockDim.x*gridDim.x) 

#define CUDA_CHECK_ERROR() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(-1); \
    } \
}

const int N = 1000005, K = 20;

__device__ ull Keys[1<<K]; 
__device__ ull Vals[1<<K];
ull Ans[N];

__device__ unsigned hash_val(ull key) {
    return key&((1<<K)-1);
}

__device__ void insert1(ull key, ull ptr) {
    unsigned hs = hash_val(key);
    while (1) {
        ull now = atomicCAS(&Keys[hs], 0, key);
        if (!now || now == key) {
            Vals[hs] = ptr;
            return;
        }
        hs = (hs+1)&((1<<K)-1);
    }
}

__global__ void insert_kernel(ull *d_keys, ull *d_ptr, int cnt) {
    LOOP(i, cnt) {
        insert1(d_keys[i], *d_ptr+i*sizeof(T));
    }
}

void insert(ull *h_keys, T *h_ptr, int cnt) {
    ull *d_keys; ull *d_ptr;
    ull num = (ull)h_ptr;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ptr, sizeof(ull));
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, &num, sizeof(ull), hipMemcpyHostToDevice);  
    insert_kernel<<<64,64>>>(d_keys, d_ptr, cnt);
    hipFree(d_keys); hipFree(d_ptr);
}

__device__ ull query1(ull key) {
    unsigned hs = hash_val(key);
    while (1) {
        if (Keys[hs] == key) return Vals[hs];
        else if (!Keys[hs]) return NULL;
        hs = (hs+1)&((1<<K)-1);
    }
}
__global__ void query_kernel(ull *d_keys, ull *d_ans, int cnt) {
    LOOP(i, cnt) {
        d_ans[i] = query1(d_keys[i]);
    }
}
void query(ull *h_keys, ull *h_ans, int cnt) {
    reverse(h_keys, h_keys+cnt);
    ull *d_keys; ull *d_ans;
    hipMalloc(&d_keys, sizeof(ull)*cnt);
    hipMalloc(&d_ans, sizeof(ull)*cnt);
    hipMemcpy(d_keys, h_keys, sizeof(ull)*cnt, hipMemcpyHostToDevice);
    query_kernel<<<64,64>>>(d_keys, d_ans, cnt);
    hipMemcpy(h_ans, d_ans, sizeof(ull)*cnt, hipMemcpyDeviceToHost);
    hipFree(d_keys); hipFree(d_ans);
}

int main() {
    data_loader<ull> data_Keys("../part_0.keys");
    data_loader<T> data_Vals("../part_0.vals");

    insert(data_Keys.data(), data_Vals.data(), data_Keys.count());
    query(data_Keys.data(), Ans, data_Keys.count());
    int cnt = data_Keys.count();
    printf("%d\n", cnt);
    for (int i = 0; i < cnt; i++) if (Ans[i]) {
        for (int j = 0; j < 64; j++) printf("%f ", ((T*)Ans[i])->v[j]);
        puts("");
    }
    return 0;
}