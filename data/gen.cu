#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cstdio>
using namespace std;
typedef unsigned long long ull;

const int N = 1000000;

hiprandGenerator_t gen;
ull keys[N+5];
float vals[N<<6|5];

void Init() {
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
    
}

int main() {
    Init();
    int n = 100000000;
    FILE *F1 = fopen("./my2.keys", "w");
    for (int i = 0; i < n; i += N) {
        hiprandGenerateLongLong(gen, keys, N);
        fwrite(keys, 1, N*sizeof(ull), F1);
    }
    fclose(F1);
    FILE *F2 = fopen("./my2.vals", "w");
    for (int i = 0; i < n; i += N) {
        hiprandGenerateUniform(gen, vals, N<<6);
        fwrite(vals, 1, (N<<6)*sizeof(float), F2);
    }
    return 0;
}